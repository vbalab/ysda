
#include <hip/hip_runtime.h>
#include <stdexcept>

void CudaCheck(hipError_t cuda_error) {
  if (cuda_error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(cuda_error));
  }
}

constexpr size_t kThreads = 1024;
